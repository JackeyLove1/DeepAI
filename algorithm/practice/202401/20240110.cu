
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
__global__ void hello_from_gpu() {
    printf("Hello World from the the GPU\n");
    printf("How are you ?\n");
}


int main(void) {
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    std::cout << "Hello, World!" << "\n";
    return 0;
}
