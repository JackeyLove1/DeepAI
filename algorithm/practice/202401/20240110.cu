
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>

__global__ void hello_from_gpu() {
    printf("Hello World from the the GPU\n");
    printf("How are you ?\n");
    const auto bid = blockIdx.x;
    const auto tid = threadIdx.x;
    const auto idx = tid + bid * blockDim.x;
    printf("bid:%d, tid:%d, idx:%d\n", bid, tid, idx);
}


int main(void) {
    hello_from_gpu<<<2, 4>>>();
    hipDeviceSynchronize();
    std::cout << "Hello, World!" << "\n";
    return 0;
}
